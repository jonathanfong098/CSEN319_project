#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include "helpers.cu"

// Initialize weights in parallel
__global__ void initialize_weights(
    double *weights,   // Weights to be initialized
    int total_weights, // Total number of weights in the network
    double std_dev,    // Standard deviation for the random weights
    unsigned long seed // Seed for random number generation
)
{
    // Index of the current weight this thread is responsible for
    int weight = blockIdx.x * blockDim.x + threadIdx.x;

    // Generate a random weight scaled by standard deviation and add a small constant to avoid zero values.
    if (weight < total_weights)
    {
        hiprandState state;
        hiprand_init(seed, weight, 0, &state);
        weights[weight] = hiprand_uniform(&state) + 0.00001;
    }
}

void initialize_network_cuda(
    int input_size,     // Number of inputs for the first layer
    int *layer_sizes,   // Sizes of each layer
    int total_layers,   // Total number of layers
    double **d_weights, // Device pointer to weights
    int *total_weights  // Output: total number of weights in the network
)
{
    int num_weights = 0;

    // Calculate the total number of weights in the network
    for (int layer = 0; layer < total_layers; layer++)
    {
        int total_neurons = layer_sizes[layer];
        int prev_layer_size;

        if (layer == 0)
        {
            prev_layer_size = input_size;
        }
        else
        {
            prev_layer_size = layer_sizes[layer - 1];
        }

        num_weights += total_neurons * (prev_layer_size + 1); // +1 for biases
    }

    hipMalloc(d_weights, num_weights * sizeof(double));

    // Initialize weights in parallel
    unsigned long seed = time(NULL);
    dim3 blockSize(256);
    dim3 gridSize((num_weights + blockSize.x - 1) / blockSize.x);
    initialize_weights<<<gridSize, blockSize>>>(*d_weights, num_weights, 0.001, seed);

    *total_weights = num_weights;
}

// Forward proprogration of a single layer
__global__ void forward_propagate(
    double *inputs,         // Input data for the current layer
    double *weights,        // Weights for the current layer
    double *outputs,        // Outputs for the current layer
    int current_input_size, // Number of inputs for each neuron
    int current_layer_size, // Number of neurons in the current layer
    int is_output_layer,    // Flag to indicate output layer
    int task                // Task type: 0 for regression, 1 for classification
)
{
    // Index of the current neuron this thread is responsible for
    int current_neuron = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the current neuron is in the current layer
    if (current_neuron < current_layer_size)
    {
        double total = 0.0;

        // Compute the weighted sum of the inputs
        for (int i = 0; i <= current_input_size; i++)
        {
            if (i < current_input_size)
            {
                total += weights[current_neuron * (current_input_size + 1) + i] * inputs[i];
            }
            else // Bias
            {
                total += weights[current_neuron * (current_input_size + 1) + i];
            }
        }

        // Apply the activation function if not the output layer
        if (is_output_layer)
        {
            if (task == 0)
            {
                outputs[current_neuron] = linear(total);
            }
            else
            {
                outputs[current_neuron] = relu(total);
            }
        }
        else
        {
            outputs[current_neuron] = relu(total);
        }
    }
}

// Backward propagation of a single layer
__global__ void backward_propagate(
    double *errors,         // Error signals for the current layer
    double *outputs,        // Outputs of the current layer
    double *weights,        // Weights of the current layer
    double *gradients,      // Gradients to be computed
    int current_input_size, // Number of inputs for each neuron
    int current_layer_size, // Number of neurons in the current layer
    int is_output_layer,    // Flag to indicate output layer
    int task                // Task type: 0 for regression, 1 for classification
)
{
    // Index of the neuron this thread is responsible for
    int current_neuron = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the neuron is in the current layer
    if (current_neuron < current_layer_size)
    {

        double gradient;

        // Compute the gradient for the current neuron
        if (is_output_layer)
        {
            printf("Output Layer\n");
            if (task == 0) // Regression
            {
                gradient = mse_loss_derivative(outputs[current_neuron], errors[current_neuron]);
                printf("MSE Loss Gradient: %f\n", gradient);
            }
        }
        else
        {
            gradient = errors[current_neuron] * relu_derivative(outputs[current_neuron]);
        }

        //  // Debug statement to print the gradient
        // if (current_neuron < 10) // Limit to first 10 neurons for readability
        // {
        //     printf("Neuron %d, Gradient: %f\n", current_neuron, gradient);
        // }

        // Update gradients for both input weights and the bias term
        for (int i = 0; i <= current_input_size; i++)
        {
            if (i < current_input_size)
            {
                gradients[current_neuron * (current_input_size + 1) + i] = gradient;
            }
            else // Bias term
            {
                gradients[current_neuron * (current_input_size + 1) + i] = gradient;
            }
        }
    }
}

// Update all weights in the network using the computed gradients
__global__ void update_weights(
    double *weights,   // Weights to be updated
    double *gradients, // Gradients to be used for the update
    double learning_rate,
    int total_weights // Total number of weights in the network
)
{
    // Index of the current weight this thread is responsible for
    int weight = blockIdx.x * blockDim.x + threadIdx.x;

    // Update each weight in the network using the corresponding gradient
    if (weight < total_weights)
    {
        weights[weight] -= learning_rate * gradients[weight];
    }
}

#if defined(TEST_CUDA)
void train_network_cuda(
    double *d_weights,    // Pre-initialized device weights
    double *h_dataset,    // Host dataset
    double *h_targets,    // Host targets
    int *layer_sizes,     // Layer sizes
    int num_samples,      // Number of training samples
    int input_size,       // Input size
    int num_layers,       // Number of layers
    int total_weights,    // Total number of weights
    int epochs,           // Number of training epochs
    double learning_rate, // Learning rate
    int task              // Task type: 0 for regression, 1 for classification
)
{
    // Timing events
    hipEvent_t start_init, stop_init, start_fp, stop_fp, start_bp, stop_bp, start_update, stop_update, start_total, stop_total;
    hipEventCreate(&start_init);
    hipEventCreate(&stop_init);
    hipEventCreate(&start_fp);
    hipEventCreate(&stop_fp);
    hipEventCreate(&start_bp);
    hipEventCreate(&stop_bp);
    hipEventCreate(&start_update);
    hipEventCreate(&stop_update);
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);

    // Total training process start
    hipEventRecord(start_total);

    // Allocate memory for inputs, outputs, errors, and gradients
    hipEventRecord(start_init);
    double *d_inputs, *d_outputs, *d_errors, *d_gradients;
    hipMalloc(&d_inputs, input_size * sizeof(double));
    hipMalloc(&d_outputs, layer_sizes[num_layers - 1] * sizeof(double));
    hipMalloc(&d_errors, layer_sizes[num_layers - 1] * sizeof(double));
    hipMalloc(&d_gradients, total_weights * sizeof(double));
    double *h_outputs = (double *)malloc(layer_sizes[num_layers - 1] * sizeof(double));
    hipEventRecord(stop_init);
    hipEventSynchronize(stop_init);

    for (int epoch = 0; epoch < epochs; epoch++)
    {
        double total_loss = 0.0;

        for (int sample_idx = 0; sample_idx < num_samples; sample_idx++)
        {
            // Forward propagation timing start
            hipEventRecord(start_fp);

            // Copy the current input to the first layer
            hipMemcpy(d_inputs, &h_dataset[sample_idx * input_size], input_size * sizeof(double), hipMemcpyHostToDevice);

            // Forward propagation through the layers
            for (int layer_idx = 0; layer_idx < num_layers; layer_idx++)
            {
                int current_input_size = (layer_idx == 0) ? input_size : layer_sizes[layer_idx - 1];
                int current_layer_size = layer_sizes[layer_idx];
                int is_output_layer = (layer_idx == num_layers - 1);

                forward_propagate<<<(current_layer_size + 255) / 256, 256>>>(
                    d_inputs, d_weights, d_outputs, current_input_size, current_layer_size, is_output_layer, task);

                hipMemcpy(d_inputs, d_outputs, current_layer_size * sizeof(double), hipMemcpyDeviceToDevice);
            }

            hipEventRecord(stop_fp);
            hipEventSynchronize(stop_fp);

            // Compute the error for the output layer
            hipMemcpy(d_errors, &h_targets[sample_idx * layer_sizes[num_layers - 1]],
                       layer_sizes[num_layers - 1] * sizeof(double), hipMemcpyHostToDevice);

            if (task == 0) // Regression task
            {
                double loss = 0.0;
                hipMemcpy(h_outputs, d_outputs, layer_sizes[num_layers - 1] * sizeof(double), hipMemcpyDeviceToHost);
                for (int i = 0; i < layer_sizes[num_layers - 1]; i++)
                {
                    double prediction = h_outputs[i];
                    double target = h_targets[sample_idx * layer_sizes[num_layers - 1] + i];
                    loss += 0.5 * pow(prediction - target, 2); // Directly calculate loss
                }
                total_loss += loss;
            }

            // Backward propagation timing start
            hipEventRecord(start_bp);

            // Backward propagation through the layers
            for (int layer_idx = num_layers - 1; layer_idx >= 0; layer_idx--)
            {
                int current_input_size = (layer_idx == 0) ? input_size : layer_sizes[layer_idx - 1];
                int current_layer_size = layer_sizes[layer_idx];
                int is_output_layer = (layer_idx == num_layers - 1);

                backward_propagate<<<(current_layer_size + 255) / 256, 256>>>(
                    d_errors, d_outputs, d_weights, d_gradients, current_input_size, current_layer_size, is_output_layer, task);
            }

            hipEventRecord(stop_bp);
            hipEventSynchronize(stop_bp);

            // Weight update timing start
            hipEventRecord(start_update);

            // Update weights
            update_weights<<<(total_weights + 255) / 256, 256>>>(d_weights, d_gradients, learning_rate, total_weights);

            hipEventRecord(stop_update);
            hipEventSynchronize(stop_update);
        }

        total_loss /= num_samples;

        // Print timing and loss
        printf("Epoch %d: Total Loss: %.6f\n", epoch, total_loss);
        // printf("Timing (ms): Init: %.3f, Forward: %.3f, Backward: %.3f, Update: %.3f, Total: %.3f\n",
        //        time_init, time_fp, time_bp, time_update, time_total);
    }
    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);

    // Timing results
    float time_init, time_fp, time_bp, time_update, time_total;
    hipEventElapsedTime(&time_init, start_init, stop_init);
    hipEventElapsedTime(&time_fp, start_fp, stop_fp);
    hipEventElapsedTime(&time_bp, start_bp, stop_bp);
    hipEventElapsedTime(&time_update, start_update, stop_update);
    hipEventElapsedTime(&time_total, start_total, stop_total);

    printf("Timing(ms:)\n  Forward: %.3f, Backward: %.3f, Update: %.3f, Total: %.3f\n", time_fp, time_bp, time_update, time_total);

    // Free resources
    hipFree(d_inputs);
    hipFree(d_outputs);
    hipFree(d_errors);
    hipFree(d_gradients);
    free(h_outputs);

    // Destroy timing events
    hipEventDestroy(start_init);
    hipEventDestroy(stop_init);
    hipEventDestroy(start_fp);
    hipEventDestroy(stop_fp);
    hipEventDestroy(start_bp);
    hipEventDestroy(stop_bp);
    hipEventDestroy(start_update);
    hipEventDestroy(stop_update);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
}
#else
void train_network_cuda(
    double *d_weights,    // Pre-initialized device weights
    double *h_dataset,    // Host dataset
    double *h_targets,    // Host targets
    int *layer_sizes,     // Layer sizes
    int num_samples,      // Number of training samples
    int input_size,       // Input size
    int num_layers,       // Number of layers
    int total_weights,    // Total number of weights
    int epochs,           // Number of training epochs
    double learning_rate, // Learning rate
    int task              // Task type: 0 for regression, 1 for classification
)
{
    // Allocate memory for inputs, outputs, errors, and gradients
    double *d_inputs, *d_outputs, *d_errors, *d_gradients;
    hipMalloc(&d_inputs, input_size * sizeof(double));
    hipMalloc(&d_outputs, layer_sizes[num_layers - 1] * sizeof(double));
    hipMalloc(&d_errors, layer_sizes[num_layers - 1] * sizeof(double));
    hipMalloc(&d_gradients, total_weights * sizeof(double));

    double *h_outputs = (double *)malloc(layer_sizes[num_layers - 1] * sizeof(double));
    for (int epoch = 0; epoch < epochs; epoch++)
    {
        double total_loss = 0.0;

        for (int sample_idx = 0; sample_idx < num_samples; sample_idx++)
        {
            // Copy the current input to the first layer
            hipMemcpy(d_inputs, &h_dataset[sample_idx * input_size], input_size * sizeof(double), hipMemcpyHostToDevice);

            // Forward propagation through the layers
            for (int layer_idx = 0; layer_idx < num_layers; layer_idx++)
            {
                int current_input_size;
                if (layer_idx == 0)
                {
                    current_input_size = input_size;
                }
                else
                {
                    current_input_size = layer_sizes[layer_idx - 1];
                }

                int current_layer_size = layer_sizes[layer_idx];
                int is_output_layer;
                if (layer_idx == num_layers - 1)
                {
                    is_output_layer = 1;
                }
                else
                {
                    is_output_layer = 0;
                }

                forward_propagate<<<(current_layer_size + 255) / 256, 256>>>(
                    d_inputs, d_weights, d_outputs, current_input_size, current_layer_size, is_output_layer, task);

                // Copy outputs to inputs for the next layer
                hipMemcpy(d_inputs, d_outputs, current_layer_size * sizeof(double), hipMemcpyDeviceToDevice);
            }

            // Compute the error for the output layer
            hipMemcpy(d_errors, &h_targets[sample_idx * layer_sizes[num_layers - 1]],
                       layer_sizes[num_layers - 1] * sizeof(double), hipMemcpyHostToDevice);

            if (task == 0) // Regression task
            {
                double loss = 0.0;
                hipMemcpy(h_outputs, d_outputs, layer_sizes[num_layers - 1] * sizeof(double), hipMemcpyDeviceToHost);
                for (int i = 0; i < layer_sizes[num_layers - 1]; i++)
                {
                    double prediction = h_outputs[i];
                    double target = h_targets[sample_idx * layer_sizes[num_layers - 1] + i];
                    loss += mse_loss(prediction, target);
                }
                total_loss += loss;
            }
            // Backward propagation through the layers
            for (int layer_idx = num_layers - 1; layer_idx >= 0; layer_idx--)
            {
                int current_input_size;
                if (layer_idx == 0)
                {
                    current_input_size = input_size;
                }
                else
                {
                    current_input_size = layer_sizes[layer_idx - 1];
                }

                int current_layer_size = layer_sizes[layer_idx];
                int is_output_layer;
                if (layer_idx == num_layers - 1)
                {
                    is_output_layer = 1;
                }
                else
                {
                    is_output_layer = 0;
                }

                backward_propagate<<<(current_layer_size + 255) / 256, 256>>>(
                    d_errors, d_outputs, d_weights, d_gradients, current_input_size, current_layer_size, is_output_layer, task);
            }

            // Update weights
            update_weights<<<(total_weights + 255) / 256, 256>>>(d_weights, d_gradients, learning_rate, total_weights);
        }

        // Normalize total loss
        total_loss /= num_samples;

        printf("Epoch %d completed. Total Loss: %.6f\n", epoch, total_loss);
    }

    hipFree(d_inputs);
    hipFree(d_outputs);
    hipFree(d_errors);
    hipFree(d_gradients);
    free(h_outputs);
}
#endif
